#include <torch/extension.h>
#include <vector>
#include "spikeKernels.h"
#include "convKernels.h"

#define CHECK_CUDA(x) AT_ASSERTM(x.type().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) AT_ASSERTM(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)
#define CHECK_DEVICE(x, y) AT_ASSERTM(x.device().index() == y.device().index(), #x " and " #y " must be in same CUDA device")

// C++ Python interface

torch::Tensor getSpikesCuda(
	torch::Tensor d_u,
	const torch::Tensor& d_nu,
	const float theta,
	const float Ts)
{
	CHECK_INPUT(d_u);
	CHECK_INPUT(d_nu);

	// check if tensor are in same device
	CHECK_DEVICE(d_u, d_nu);

	auto d_s = torch::empty_like(d_u);

	// TODO implement for different data types

	// set the current cuda device to wherever the tensor d_u resides
	hipSetDevice(d_u.device().index());

	unsigned nuSize = d_nu.size(-1);
	unsigned Ns = d_u.size(-1);
	unsigned nNeurons = d_u.size(0) * d_u.size(1) * d_u.size(2) * d_u.size(3);
	getSpikes<float>(d_s.data<float>(), d_u.data<float>(), d_nu.data<float>(), nNeurons, nuSize, Ns, theta, Ts);

	return d_s;
}

torch::Tensor convCuda(torch::Tensor input, torch::Tensor filter, float Ts)
{
	CHECK_INPUT(input);
	CHECK_INPUT(filter);
	CHECK_DEVICE(input, filter);

	hipSetDevice(input.device().index());

	auto output = torch::empty_like(input);
	
	unsigned signalSize = input.size(-1); 
	unsigned filterSize = filter.numel();
	unsigned nNeurons   = input.numel()/input.size(-1); 
	conv<float>(output.data<float>(), input.data<float>(), filter.data<float>(), signalSize, filterSize, nNeurons, Ts);

	return output;
}

torch::Tensor corrCuda(torch::Tensor input, torch::Tensor filter, float Ts)
{
	CHECK_INPUT(input);
	CHECK_INPUT(filter);
	CHECK_DEVICE(input, filter);

	hipSetDevice(input.device().index());

	auto output = torch::empty_like(input);
	
	unsigned signalSize = input.size(-1); 
	unsigned filterSize = filter.numel();
	unsigned nNeurons   = input.numel()/input.size(-1); 
	corr<float>(output.data<float>(), input.data<float>(), filter.data<float>(), signalSize, filterSize, nNeurons, Ts);

	return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m)
{
	m.def("getSpikes", &getSpikesCuda, "Get spikes (CUDA)");
	m.def("conv"     , &convCuda     , "Convolution in time (CUDA)");
	m.def("corr"     , &corrCuda     , "Correlation in time (CUDA)");
}
